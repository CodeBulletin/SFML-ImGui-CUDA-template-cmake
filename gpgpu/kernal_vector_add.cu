#include "hip/hip_runtime.h"
#include "kernal/vector_add.hpp"
#include "cuda.cuh"

__global__ void kernal_vector_add(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

void kernal::vector_add(float *a, float *b, float *c, int n)
{
    int size = n * sizeof(float);
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    kernal_vector_add<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}